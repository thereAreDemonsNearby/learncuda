#include "hip/hip_runtime.h"
#include <stdio.h>

int main()
{
    int devCount;
    hipGetDeviceCount(&devCount);
    printf("device count: %d\n", devCount);
    for (int i = 0; i < devCount; ++i) {
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);
        printf("ver: %d.%d\n", devProp.major, devProp.minor);
        printf("max threads per block: %d\n", (int)devProp.maxThreadsPerBlock);
        printf("number of SMs: %d\n", (int)devProp.multiProcessorCount);
        printf("warp size: %d\n", (int)devProp.warpSize);
        printf("max warps per SM: %d\n", devProp.maxThreadsPerMultiProcessor / devProp.warpSize);
        printf("registers available per SM: %d\n", (int)devProp.regsPerBlock);
        printf("shared memory available per SM: %d\n", (int)devProp.sharedMemPerBlock);
        printf("clock frequency: %d\n", (int)devProp.clockRate);
        printf("total const memory: %d\n", (int)devProp.totalConstMem);
        printf("max threads dimBlock.x: %d\n", (int)devProp.maxThreadsDim[0]);
        printf("max threads dimBlock.y: %d\n", (int)devProp.maxThreadsDim[1]);
        printf("max threads dimBlock.z: %d\n", (int)devProp.maxThreadsDim[2]);
        printf("max blocks dimGrid.x: %d\n", (int)devProp.maxGridSize[0]);
        printf("max blocks dimGrid.y: %d\n", (int)devProp.maxGridSize[1]);
        printf("max blocks dimGrid.z: %d\n", (int)devProp.maxGridSize[2]);
        printf("memory clock rate: %d\n", (int)devProp.memoryClockRate);
        printf("memory bus width: %d\n", (int)devProp.memoryBusWidth);
        printf("memory pitch: %lld\n", devProp.memoryBusWidth);
        printf("\n");        
    }
}
